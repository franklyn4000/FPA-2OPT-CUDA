#include "hip/hip_runtime.h"
//
// Created by franklyn on 4/11/24.
//

#include "fpa_cuda.cuh"
#include <iostream>
#include <random>
#include <math.h>
#include "omp.h"
#include "../iolib.cuh"
#include "../initialSolutionGenerator.cuh"

#include "../utils.cuh"

#include "../parallel/twoOpt_parallel.h"
#include "../parallel/fitnessComputer_parallel.h"
#include "../parallel/twoOpt_parallel.h"
#include "../parallel/pollinator_parallel.h"
#include "../objects/paths.h"
#include "pathSmoother_cuda.cuh"

#define CHECK_CUDA(call)                                            \
{                                                                   \
const hipError_t error = call;                                     \
if (error != hipSuccess)                                           \
{                                                                   \
printf("ERROR:: File: %s, Line: %d, ", __FILE__, __LINE__);         \
printf("code: %d, reason: %s\n", error, hipGetErrorString(error)); \
exit(EXIT_FAILURE);                                                 \
}                                                                   \
}

void computeFPA_cuda(
        Config &config, Drone &drone, InitialConditions &init) {

    float a_utopia = drone.min_altitude;
    float f_utopia = calculateFUtopia(init);

    double pollination_start_time = 0;
    double pollination_time_taken = 0;

    double twoopt_start_time = 0;
    double twoopt_time_taken = 0;

    float nwp = 0;

    int max_waypoints = config.path_length * 3;

    Paths_cuda paths;
    paths.bestFitness = -1.0f;

    dim3 dimBlock(256);
    dim3 dimGrid((config.population + dimBlock.x - 1) / dimBlock.x);

    //Initialize a population of n flowers/pollen gametes with random solutions
    // paths.rawPaths = generateSolutions(init,
    //                                    config.path_length, config.population);

    //   paths.pollinatedPaths = generateSolutions(init,
    //                                            config.path_length, config.population);


    float* hostPtr = new float[config.population * max_waypoints];

    paths.rawPaths.width = max_waypoints;
    paths.rawPaths.height = config.population;
    size_t paths_size = paths.rawPaths.width * paths.rawPaths.height * sizeof(float);
    printf("size %i \n", paths_size);

    for (int i = 0; i < paths.rawPaths.height; i++)
        for (int j = 0; j < paths.rawPaths.width; j++) {
            hostPtr[i * paths.rawPaths.width + j] = j + 1.0;
           // printf("row %i column %i value %f \n", i, j, hostPtr[Ncols * j + i] );
        }
/*
    hipMallocPitch(&paths.rawPaths, &pitch, Ncols * sizeof(float), Nrows);
    hipMemcpy2D(&paths.rawPaths, pitch, hostPtr, Ncols*sizeof(float), Ncols*sizeof(float), Nrows, hipMemcpyHostToDevice);
*/

size_t pitch = 0;

    hipMalloc(&paths.rawPaths.elements, paths_size);
    hipMemcpy(paths.rawPaths.elements, hostPtr, paths_size,
               hipMemcpyHostToDevice);

    //  CHECK_CUDA(hipMalloc((void **) &paths.rawPaths, config.path_length * 3 * config.population * sizeof(float)));
    //  CHECK_CUDA(hipMalloc((void **) &paths.pollinatedPaths, config.path_length * 3 * config.population * sizeof(float)));
    CHECK_CUDA(hipMalloc((void **) &paths.fitnesses, config.population * sizeof(float)));
    CHECK_CUDA(hipMalloc((void **) &paths.N_wps, config.population * sizeof(float)));

/*
    float** rawPaths;

    //need to do malloc2D stuff

    hipHostMalloc(&rawPaths, config.path_length * 3 * config.population * sizeof(float));
    if (rawPaths == NULL) {
        fprintf(stderr, "amemory allocation failed!\n");
        return;
    }

    rawPaths = generateSolutions_cuda(init,config.path_length, config.population);

    hipMemcpy(paths.rawPaths, rawPaths, config.path_length * 3 * config.population * sizeof(float), hipMemcpyHostToDevice);

*/
    float *test_Nwps;
    hipHostMalloc(&test_Nwps, config.population * sizeof(float));

    for (int i = 0; i < config.population; i++) {
        test_Nwps[i] = i + 0.0;
    }
    hipMemcpy(paths.fitnesses, test_Nwps, config.population * sizeof(float), hipMemcpyHostToDevice);


    double smoothing_start_time = omp_get_wtime();

    smoothPaths_cuda<<<dimGrid, dimBlock>>>(paths, drone.turn_radius, drone.turn_radius * 2, pitch);
    hipDeviceSynchronize();

    double smoothing_time_taken = omp_get_wtime() - smoothing_start_time;


    double fitness_start_time = omp_get_wtime();

    //computeFitnesses(paths, config.heightMap, drone.max_asc_angle, drone.max_desc_angle, a_utopia, f_utopia,
    //                 config.resolution);

    double fitness_time_taken = omp_get_wtime() - fitness_start_time;


    printf("Iteration: ");
    for (int i = 0; i < config.iter_max; i++) {

        pollination_start_time = omp_get_wtime();

        //pollinate_parallel(paths, config.p_switch);

        pollination_time_taken += omp_get_wtime() - pollination_start_time;

        smoothing_start_time = omp_get_wtime();

        //  smoothPaths_cuda(paths, drone.turn_radius, drone.turn_radius * 2);

        smoothing_time_taken += omp_get_wtime() - smoothing_start_time;

        fitness_start_time = omp_get_wtime();

        //computeFitnesses(paths, config.heightMap, drone.max_asc_angle, drone.max_desc_angle, a_utopia, f_utopia,
        //               config.resolution);

        fitness_time_taken += omp_get_wtime() - fitness_start_time;


        int quarter = std::ceil(config.iter_max / 4.0);
        int half = std::ceil(config.iter_max / 2.0);
        int eight = std::ceil(config.iter_max / 8.0);
/*
        if (i == eight) {
            std::vector<float> smoothedPath = smoothPath(
                    paths.fittestPath,
                    drone.turn_radius, drone.turn_radius * 2, nwp);
            save_to_csv(smoothedPath, "../heightMapper/fittest1.csv");
        } else if (i == quarter) {
            std::vector<float> smoothedPath = smoothPath(
                    paths.fittestPath,
                    drone.turn_radius, drone.turn_radius * 2, nwp);
            save_to_csv(smoothedPath, "../heightMapper/fittest2.csv");
        } else if (i == half) {
            std::vector<float> smoothedPath = smoothPath(
                    paths.fittestPath,
                    drone.turn_radius, drone.turn_radius * 2, nwp);
            save_to_csv(smoothedPath, "../heightMapper/fittest3.csv");
        }
*/

        /*if (i % config.two_opt_freq == 0) {
            printf("%i ", i);
            twoopt_start_time = omp_get_wtime();

            twoOptParallel(paths, drone.turn_radius, drone.turn_radius * 2, config.heightMap, drone.max_asc_angle,
                           drone.max_desc_angle, a_utopia, f_utopia, config.resolution);

            twoopt_time_taken += omp_get_wtime() - twoopt_start_time;

            computeBestFitness(paths);
        }*/
        //   computeBestFitness(paths);

    }
    printf("\n");
/*
    std::vector<float> smoothedPath = smoothPath_cuda(
            paths.fittestPath,
            drone.turn_radius, drone.turn_radius * 2, nwp);

    save_to_csv(smoothedPath, "../heightMapper/fittest4.csv");
*/
    double totalTime = pollination_time_taken + smoothing_time_taken + fitness_time_taken + twoopt_time_taken;

    printf("\nPollination, Smoothing, Fitness, 2-opt:\n%.2f, %.2f, %.2f, %.2f\n", pollination_time_taken / totalTime,
           smoothing_time_taken / totalTime, fitness_time_taken / totalTime, twoopt_time_taken / totalTime);

    printf("CUDA Algorithm time: %f Reached Fitness: %f\n", totalTime, paths.bestFitness);


    float *res;

    hipHostMalloc(&res, config.population * sizeof(float));
    if (res == NULL) {
        fprintf(stderr, "bmemory allocation failed!\n");
        return;
    }

    hipMemcpy(res, paths.N_wps, config.population * sizeof(float), hipMemcpyDeviceToHost);


    for (int i = 0; i < config.population; i++) {
        //printf("%f  ", rawPaths[i][5]);
        if(res[i] > 0.0){
          //  printf("%f\n", res[i]);
        }

    }

/*
    std::vector<float> testPath;

    testPath.push_back(0);
    testPath.push_back(0);
    testPath.push_back(550);

    testPath.push_back(0);
    testPath.push_back(120);
    testPath.push_back(552);

    testPath.push_back(0);
    testPath.push_back(300);
    testPath.push_back(400);

    testPath.push_back(400);
    testPath.push_back(300);
    testPath.push_back(800);

    testPath.push_back(0);
    testPath.push_back(50);
    testPath.push_back(500);

    testPath.push_back(600);
    testPath.push_back(600);
    testPath.push_back(200);

    double test_start_time;
    double test_time_taken = 0;

    std::vector<float> testSmoothed = smoothPath(
            testPath,
            100, 400, nwp);

    for (int i = 0; i < 25000; i++) {
        test_start_time = omp_get_wtime();

        float F =
                computeFitness(testSmoothed,
                               config.heightMap,
                               nwp,
                               drone.max_asc_angle,
                               drone.max_desc_angle,
                               a_utopia, f_utopia, config.resolution);

        test_time_taken += omp_get_wtime() - test_start_time;
    }





    printf("Test Path Smoothing: %f\n", test_time_taken);
*/


    delete[] hostPtr;
}

