#include "hip/hip_runtime.h"
//
// Created by franklyn on 6/17/24.
//

#include "pollinator_cuda.cuh"
#include <iostream>

__global__ void setup_curand_kernel(hiprandStatePhilox4_32_10_t *state) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    /* Each thread gets same seed, a different sequence
       number, no offset */
    hiprand_init(1234, idx, 0, &state[idx]);
}

__global__ void pollinate_cuda(
        Paths_cuda paths,
        float p_switch,
        int heightMapWidth,
        int heightMapHeight,
        hiprandStatePhilox4_32_10_t *hiprandState) {

    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if (idx < paths.rawPaths.n_paths) {
        hiprandStatePhilox4_32_10_t localState = hiprandState[idx];

        int n = paths.rawPaths.n_waypoints;

        float4 dis = hiprand_uniform4(&localState);



        // printf("%f\n", dis.w);

        int pathIndex = idx * paths.rawPaths.n_waypoints * 3;

        if (dis.w < p_switch) {
            //global pollinate

            // TODO precompute large array of levy numbers?
            //float *L = levy_p(paths.rawPaths.n_waypoints * 3, dis.x);
            //  float gamma = 0.1;

            paths.pollinatedPaths.elements[pathIndex + 3 * 0] = paths.rawPaths.elements[pathIndex + 3 * 0];
            paths.pollinatedPaths.elements[pathIndex + 3 * 0 + 1] = paths.rawPaths.elements[pathIndex + 3 * 0 + 1];
            paths.pollinatedPaths.elements[pathIndex + 3 * 0 + 2] = paths.rawPaths.elements[pathIndex + 3 * 0 + 2];
            paths.pollinatedPaths.elements[pathIndex + 3 * paths.rawPaths.n_waypoints - 3] = paths.rawPaths.elements[
                    pathIndex + 3 * paths.rawPaths.n_waypoints - 3];
            paths.pollinatedPaths.elements[pathIndex + 3 * paths.rawPaths.n_waypoints - 2] = paths.rawPaths.elements[
                    pathIndex + 3 * paths.rawPaths.n_waypoints - 2];
            paths.pollinatedPaths.elements[pathIndex + 3 * paths.rawPaths.n_waypoints - 1] = paths.rawPaths.elements[
                    pathIndex + 3 * paths.rawPaths.n_waypoints - 1];


            for (int i = 1; i < paths.rawPaths.n_waypoints - 1; i++) {

                float coord = paths.rawPaths.elements[pathIndex + 3 * i + 0];
                float coord1 = paths.rawPaths.elements[pathIndex + 3 * i + 1];
                float coord2 = paths.rawPaths.elements[pathIndex + 3 * i + 2];
                //   paths.pollinatedPaths.elements[pathIndex + 3 * i] = paths.rawPaths.elements[pathIndex + 3 * i];
                //   paths.pollinatedPaths.elements[pathIndex + 3 * i + 1] = paths.rawPaths.elements[pathIndex + 3 * i + 1];
                //     paths.pollinatedPaths.elements[pathIndex + 3 * i + 2] = paths.rawPaths.elements[pathIndex + 3 * i + 2];

                float boundedCoord = max(min(coord, (float)heightMapWidth - 1), 0.0f);
                float boundedCoord1 = max(min(coord1, (float)heightMapHeight - 1), 0.0f);
                float boundedCoord2 = max(min(coord2, 2799.9f), 1900.0f);

                paths.pollinatedPaths.elements[pathIndex + 3 * i + 0] =
                        boundedCoord + dis.z * (paths.fittestPath[3 * i + 0] - boundedCoord);
                paths.pollinatedPaths.elements[pathIndex + 3 * i + 1] =
                        boundedCoord1 + dis.z * (paths.fittestPath[3 * i + 1] - boundedCoord1);
                paths.pollinatedPaths.elements[pathIndex + 3 * i + 2] =
                        boundedCoord2 + dis.z * (paths.fittestPath[3 * i + 2] - boundedCoord2);

                // paths.pollinatedPaths[pathIndex][3 * i + 0] =
                //         coord + L[3 * i] * (paths.fittestPath[3 * i + k] - coord);


            }

        } else {
            int j = __float2int_rd(dis.y * (paths.rawPaths.n_waypoints * 3 - 1));
            int l = __float2int_rd(dis.z * (paths.rawPaths.n_waypoints * 3 - 1));

            paths.pollinatedPaths.elements[pathIndex + 3 * 0] = paths.rawPaths.elements[pathIndex + 3 * 0];
            paths.pollinatedPaths.elements[pathIndex + 3 * 0 + 1] = paths.rawPaths.elements[pathIndex + 3 * 0 + 1];
            paths.pollinatedPaths.elements[pathIndex + 3 * 0 + 2] = paths.rawPaths.elements[pathIndex + 3 * 0 + 2];

            for (int i = 1; i < paths.rawPaths.n_waypoints - 1; i++) {
                float inc = paths.rawPaths.elements[pathIndex + 3 * i + 0] + dis.x * (paths.rawPaths.elements[j + 3 * i + 0] - paths.rawPaths.elements[l + 3 * i + 0]);
                float boundedInc = max(min(inc, (float)heightMapWidth - 1), 0.0f);
                paths.pollinatedPaths.elements[pathIndex + 3 * i + 0] = boundedInc;

                inc = paths.rawPaths.elements[pathIndex + 3 * i + 1] + dis.x * (paths.rawPaths.elements[j + 3 * i + 1] - paths.rawPaths.elements[l + 3 * i + 1]);
                boundedInc = max(min(inc, (float)heightMapHeight - 1), 0.0f);
                paths.pollinatedPaths.elements[pathIndex + 3 * i + 1] = boundedInc;

                inc = paths.rawPaths.elements[pathIndex + 3 * i + 2] + dis.x * (paths.rawPaths.elements[j + 3 * i + 2] - paths.rawPaths.elements[l + 3 * i + 2]);
                boundedInc = max(min(inc, 2799.9f), 1900.0f);
                paths.pollinatedPaths.elements[pathIndex + 3 * i + 2] = boundedInc;

                //  paths.pollinatedPaths.elements[0] = 1.0;// + dis.x * (paths.rawPaths.elements[j + 3 * i + k] - paths.rawPaths.elements[k + 3 * i + k]);}
            }

            paths.pollinatedPaths.elements[pathIndex + 3 * paths.rawPaths.n_waypoints - 3] = paths.rawPaths.elements[
                    pathIndex + 3 * paths.rawPaths.n_waypoints - 3];
            paths.pollinatedPaths.elements[pathIndex + 3 * paths.rawPaths.n_waypoints - 2] = paths.rawPaths.elements[
                    pathIndex + 3 * paths.rawPaths.n_waypoints - 2];
            paths.pollinatedPaths.elements[pathIndex + 3 * paths.rawPaths.n_waypoints - 1] = paths.rawPaths.elements[
                    pathIndex + 3 * paths.rawPaths.n_waypoints - 1];

        }


        //  paths.rawPaths = paths.pollinatedPaths;
        hiprandState[idx] = localState;
    }
}