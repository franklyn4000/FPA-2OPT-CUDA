#include "hip/hip_runtime.h"
//
// Created by franklyn on 6/17/24.
//

#include "pollinator_cuda.cuh"
#include <iostream>

__global__ void setup_curand_kernel(hiprandStatePhilox4_32_10_t *state) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    /* Each thread gets same seed, a different sequence
       number, no offset */
    hiprand_init(1234, idx, 0, &state[idx]);
}



__device__ float4 levy_device(hiprandStatePhilox4_32_10_t localState, float sig_p, float oneOverBeta) {
    float4 levy;

    float4 norm = hiprand_normal4(&localState);
    float2 norm2 = hiprand_normal2(&localState);

    levy.x = abs(0.2 * norm.x * sig_p / pow(abs(norm.y), oneOverBeta));
    levy.y = abs(0.2 * norm.w * sig_p / pow(abs(norm.z), oneOverBeta));
    levy.z = abs(0.2 * norm2.x * sig_p / pow(abs(norm2.y), oneOverBeta));


    return levy;
}

__global__ void pollinate_cuda(
        Paths_cuda paths,
        float p_switch,
        int heightMapWidth,
        int heightMapHeight,
        hiprandStatePhilox4_32_10_t *hiprandState) {

    int idx = threadIdx.x + blockIdx.x * blockDim.x;


    float beta = 1.5;
    float oneOverBeta = 1.0 / beta;
    float sig_p = pow(tgamma(1.0 + beta) * sin(M_PI * beta / 2.0) /
                              (tgamma((1.0 + beta) / 2.0) * beta * pow(2.0, (beta - 1.0) / 2.0)), oneOverBeta);

    if (idx < paths.rawPaths.n_paths) {
        hiprandStatePhilox4_32_10_t localState = hiprandState[idx];

        int n = paths.rawPaths.n_waypoints;

        float4 dis = hiprand_uniform4(&localState);



        // printf("%f\n", dis.w);

        int pathIndex = idx * paths.rawPaths.n_waypoints * 3;

        if (dis.w < p_switch) {
            //global pollinate

            // TODO precompute large array of levy numbers?
           float4 L = levy_device(localState, sig_p, oneOverBeta);
            //  float gamma = 0.1;

            paths.pollinatedPaths.elements[pathIndex + 3 * 0] = paths.rawPaths.elements[pathIndex + 3 * 0];
            paths.pollinatedPaths.elements[pathIndex + 3 * 0 + 1] = paths.rawPaths.elements[pathIndex + 3 * 0 + 1];
            paths.pollinatedPaths.elements[pathIndex + 3 * 0 + 2] = paths.rawPaths.elements[pathIndex + 3 * 0 + 2];
            paths.pollinatedPaths.elements[pathIndex + 3 * paths.rawPaths.n_waypoints - 3] = paths.rawPaths.elements[
                    pathIndex + 3 * paths.rawPaths.n_waypoints - 3];
            paths.pollinatedPaths.elements[pathIndex + 3 * paths.rawPaths.n_waypoints - 2] = paths.rawPaths.elements[
                    pathIndex + 3 * paths.rawPaths.n_waypoints - 2];
            paths.pollinatedPaths.elements[pathIndex + 3 * paths.rawPaths.n_waypoints - 1] = paths.rawPaths.elements[
                    pathIndex + 3 * paths.rawPaths.n_waypoints - 1];


            for (int i = 1; i < paths.rawPaths.n_waypoints - 1; i++) {

                float coord = paths.rawPaths.elements[pathIndex + 3 * i + 0];
                float coord1 = paths.rawPaths.elements[pathIndex + 3 * i + 1];
                float coord2 = paths.rawPaths.elements[pathIndex + 3 * i + 2];
                //   paths.pollinatedPaths.elements[pathIndex + 3 * i] = paths.rawPaths.elements[pathIndex + 3 * i];
                //   paths.pollinatedPaths.elements[pathIndex + 3 * i + 1] = paths.rawPaths.elements[pathIndex + 3 * i + 1];
                //     paths.pollinatedPaths.elements[pathIndex + 3 * i + 2] = paths.rawPaths.elements[pathIndex + 3 * i + 2];

                float val = coord + L.x * (paths.fittestPath[3 * i + 0] - coord);
                float val1 = coord1 + L.y * (paths.fittestPath[3 * i + 1] - coord1);
                float val2 = coord2 + L.z * (paths.fittestPath[3 * i + 2] - coord2);

                float bounded = max(min(val, (float) heightMapWidth - 1), 0.0f);
                float bounded1 = max(min(val1, (float) heightMapHeight - 1), 0.0f);
                float bounded2 = max(min(val2, 2799.9f), 1900.0f);

                paths.pollinatedPaths.elements[pathIndex + 3 * i + 0] = bounded;

                paths.pollinatedPaths.elements[pathIndex + 3 * i + 1] = bounded1;
                paths.pollinatedPaths.elements[pathIndex + 3 * i + 2] = bounded2;

                // paths.pollinatedPaths[pathIndex][3 * i + 0] =
                //         coord + L[3 * i] * (paths.fittestPath[3 * i + k] - coord);


            }

          //  free(L);
        } else {
            int j = __float2int_rd(dis.y * (paths.rawPaths.n_waypoints * 3 - 1));
            int l = __float2int_rd(dis.z * (paths.rawPaths.n_waypoints * 3 - 1));

            paths.pollinatedPaths.elements[pathIndex + 3 * 0] = paths.rawPaths.elements[pathIndex + 3 * 0];
            paths.pollinatedPaths.elements[pathIndex + 3 * 0 + 1] = paths.rawPaths.elements[pathIndex + 3 * 0 + 1];
            paths.pollinatedPaths.elements[pathIndex + 3 * 0 + 2] = paths.rawPaths.elements[pathIndex + 3 * 0 + 2];

            for (int i = 1; i < paths.rawPaths.n_waypoints - 1; i++) {
                float inc = paths.rawPaths.elements[pathIndex + 3 * i + 0] +
                            dis.x * (paths.rawPaths.elements[j + 3 * i + 0] - paths.rawPaths.elements[l + 3 * i + 0]);
                float boundedInc = max(min(inc, (float) heightMapWidth - 1), 0.0f);
                paths.pollinatedPaths.elements[pathIndex + 3 * i + 0] = boundedInc;

                inc = paths.rawPaths.elements[pathIndex + 3 * i + 1] +
                      dis.x * (paths.rawPaths.elements[j + 3 * i + 1] - paths.rawPaths.elements[l + 3 * i + 1]);
                boundedInc = max(min(inc, (float) heightMapHeight - 1), 0.0f);
                paths.pollinatedPaths.elements[pathIndex + 3 * i + 1] = boundedInc;

                inc = paths.rawPaths.elements[pathIndex + 3 * i + 2] +
                      dis.x * (paths.rawPaths.elements[j + 3 * i + 2] - paths.rawPaths.elements[l + 3 * i + 2]);
                boundedInc = max(min(inc, 2799.9f), 1900.0f);
                paths.pollinatedPaths.elements[pathIndex + 3 * i + 2] = boundedInc;

                //  paths.pollinatedPaths.elements[0] = 1.0;// + dis.x * (paths.rawPaths.elements[j + 3 * i + k] - paths.rawPaths.elements[k + 3 * i + k]);}
            }

            paths.pollinatedPaths.elements[pathIndex + 3 * paths.rawPaths.n_waypoints - 3] = paths.rawPaths.elements[
                    pathIndex + 3 * paths.rawPaths.n_waypoints - 3];
            paths.pollinatedPaths.elements[pathIndex + 3 * paths.rawPaths.n_waypoints - 2] = paths.rawPaths.elements[
                    pathIndex + 3 * paths.rawPaths.n_waypoints - 2];
            paths.pollinatedPaths.elements[pathIndex + 3 * paths.rawPaths.n_waypoints - 1] = paths.rawPaths.elements[
                    pathIndex + 3 * paths.rawPaths.n_waypoints - 1];

        }


        //  paths.rawPaths = paths.pollinatedPaths;
        hiprandState[idx] = localState;
    }
}