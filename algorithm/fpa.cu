#include "hip/hip_runtime.h"
//
// Created by franklyn on 3/17/24.
//

#include "fpa.cuh"
#include <vector>
#include <random>
#include <math.h>
#include "iolib.cuh"

std::vector<float>
generateVector(float x1, float y1, float z1, float xn, float yn, float zn, float x_min, float x_max, float y_min,
               float y_max, float z_min, float z_max) {
    // Generate a random size for the vector
    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_int_distribution<> distr(5, 5); // replace 10 with maximum possible value of n

    std::uniform_real_distribution<> distrX(x_min, x_max);
    std::uniform_real_distribution<> distrY(y_min, y_max);
    std::uniform_real_distribution<> distrZ(z_min, z_max);

    int n = distr(gen);

    std::vector<float> vector(3 * n, 0);

    // Set initial values
    vector[0] = x1;
    vector[1] = y1;
    vector[2] = z1;

    for (int i = 1; i < n - 1; ++i) {
        vector[3 * i] = distrX(gen);
        vector[3 * i + 1] = distrY(gen);
        vector[3 * i + 2] = distrZ(gen);
    }

    // Set final values
    vector[3 * n - 3] = xn;
    vector[3 * n - 2] = yn;
    vector[3 * n - 1] = zn;

    return vector;
}

std::vector <std::vector<float>> smoothPaths(std::vector <std::vector<float>> paths, float turnRadius, int n_pi) {
    std::vector <std::vector<float>> smoothedPaths;

    for (const std::vector<float> &path: paths) {
        std::vector<float> smoothedPath;

        smoothedPath.push_back(path[0]);
        smoothedPath.push_back(path[1]);
        smoothedPath.push_back(path[2]);

        int n = path.size() / 3;

        for (int i = 1; i < n - 1; i++) {
            std::vector<float> P1;
            std::vector<float> P;
            std::vector<float> P2;
            P1.push_back(path[3 * (i - 1)]);
            P1.push_back(path[3 * (i - 1) + 1]);
            P1.push_back(path[3 * (i - 1) + 2]);
            P.push_back(path[3 * i]);
            P.push_back(path[3 * i + 1]);
            P.push_back(path[3 * i + 2]);
            P2.push_back(path[3 * (i + 1)]);
            P2.push_back(path[3 * (i + 1) + 1]);
            P2.push_back(path[3 * (i + 1) + 2]);



            //unit vector from P1 to P
            std::vector<float> tau_1;
            float mag_1 = sqrt(
                    (P[0] - P1[0]) * (P[0] - P1[0]) +
                    (P[1] - P1[1]) * (P[1] - P1[1]) +
                    (P[2] - P1[2]) * (P[2] - P1[2])
            );
            tau_1.push_back((P[0] - P1[0]) / mag_1);
            tau_1.push_back((P[1] - P1[1]) / mag_1);
            tau_1.push_back((P[2] - P1[2]) / mag_1);

            //unit vector from P to P2
            std::vector<float> tau_2;
            float mag_2 = sqrt(
                    (P2[0] - P[0]) * (P2[0] - P[0]) +
                    (P2[1] - P[1]) * (P2[1] - P[1]) +
                    (P2[2] - P[2]) * (P2[2] - P[2])
            );
            tau_2.push_back((P2[0] - P[0]) / mag_2);
            tau_2.push_back((P2[1] - P[1]) / mag_2);
            tau_2.push_back((P2[2] - P[2]) / mag_2);

            float dot = tau_1[0] * tau_2[0] + tau_1[1] * tau_2[1] + tau_1[2] * tau_2[2];

            //angle alpha between the two unit vectors
            float alpha = M_PI - acos(dot);

            //compute Center C of tangent circle using C = P + turnRadius * csc(alpha) * (tau_2 - tau_1)
            std::vector<float> C(3, 0);
            for (int j = 0; j < 3; j++) {


                C[j] = P[j] + turnRadius * (1 / sin(alpha)) * (tau_2[j] - tau_1[j]);
                printf("\ncalculating C %f: %f + %f * %f * (%f - %f)  \n", C[j], P[j], turnRadius, (1 / sin(alpha)),
                       tau_2[j], tau_1[j]);
            }


            //calculate distance between P and C
            float distance_PC = sqrt(
                    (P[0] - C[0]) * (P[0] - C[0]) +
                    (P[1] - C[1]) * (P[1] - C[1]) +
                    (P[2] - C[2]) * (P[2] - C[2])
            );

            printf("\n dist1: %f dist2: %f distC: %f \n", mag_1, mag_2, distance_PC);
            if (distance_PC > min(mag_1, mag_2)) {
                //cannot smooth trajectory
                smoothedPath.push_back(P[0]);
                smoothedPath.push_back(P[1]);
                smoothedPath.push_back(P[2]);
                continue;
            }

            //compute the number n of waypoints required to draw a circular arc using n = max(3, ceil(n_pi * (pi - alpha)/pi))
            int n_waypoints = std::max(3, static_cast<int>(std::ceil(n_pi * (M_PI - alpha) / M_PI)));


            for (int j = 0; j < n_waypoints; j++) {
                //compute angle omega = j * (pi - alpha) / (n - 1)
                float omega = j * (M_PI - alpha) / (n_waypoints - 1);


                for (int k = 0; k < 3; k++) {
                    /*
                    smoothedPath.push_back(
                            C[k] -
                            turnRadius *
                            (1 / sin(alpha)) *
                            tau_1[k] *
                            cos(alpha + omega) -
                            turnRadius *
                            (1 / sin(alpha)) *
                            tau_2[k] *
                            cos(omega)
                    );
                     */
                }

                //compute point C_i = C - turnRadius * csc(alpha) * tau_1 * cos(alpha + omega) - turnRadius * csc(alpha) * tau_2 * cos(omega)
                // VERIFY

                smoothedPath.push_back(C[0] - turnRadius * (1 / sin(alpha)) * tau_1[0] * cos(alpha + omega) -
                                       turnRadius * (1 / sin(alpha)) * tau_2[0] * cos(omega));
                smoothedPath.push_back(C[1] - turnRadius * (1 / sin(alpha)) * tau_1[1] * cos(alpha + omega) -
                                       turnRadius * (1 / sin(alpha)) * tau_2[1] * cos(omega));
                smoothedPath.push_back(C[2] - turnRadius * (1 / sin(alpha)) * tau_1[2] * cos(alpha + omega) -
                                       turnRadius * (1 / sin(alpha)) * tau_2[2] * cos(omega));

            }
/*
            smoothedPath.push_back(C[0]);
            smoothedPath.push_back(C[1]);
            smoothedPath.push_back(C[2]);
            smoothedPath.push_back(P[0]);
            smoothedPath.push_back(P[1]);
            smoothedPath.push_back(P[2]);

            smoothedPath.push_back(path[3 * i]);
            smoothedPath.push_back(path[3 * i + 1]);
            smoothedPath.push_back(path[3 * i + 2]);*/
        }

        smoothedPath.push_back(path[3 * n - 3]);
        smoothedPath.push_back(path[3 * n - 2]);
        smoothedPath.push_back(path[3 * n - 1]);


        smoothedPaths.push_back(smoothedPath);
    }

    return smoothedPaths;
}

std::vector<float> computeFitnesses(std::vector <std::vector<float>> paths, const std::vector <std::vector<double>> &heightMap) {
    std::vector<float> fitnesses;

    //penalty term

    //P = d_ug + d_dz + d_ea + (N_wp_unsmoothed / N_wp * l_traj)



    float interval = 20.0f;

    for (const std::vector<float> &path: paths) {
        int n = path.size() / 3;

        for (int i = 0; i < n - 0; i++) {
            printf("\nP X: %f Y: %f Z: %f", path[3 * i], path[3 * i + 1], path[3 * i + 2]);
        }

    }


    for (const std::vector<float> &path: paths) {
        float d_ug = 0.0;
        float d_dz = 0.0;
        float d_ea = 0.0;
        float N_wp_unsmoothed = 0.0;
        float N_wp = 50.0;
        float l_traj = 0.0;

        int n = path.size() / 3;
        bool underground = false;
        bool undergroundLast = false;

        for (int i = 0; i < n - 1; i++) {
            std::vector<float> P1;
            std::vector<float> P2;
            P1.push_back(path[3 * i]);
            P1.push_back(path[3 * i + 1]);
            P1.push_back(path[3 * i + 2]);
            P2.push_back(path[3 * (i + 1)]);
            P2.push_back(path[3 * (i + 1) + 1]);
            P2.push_back(path[3 * (i + 1) + 2]);

            float distance_P1P2 = sqrt(
                    (P2[0] - P1[0]) * (P2[0] - P1[0]) +
                    (P2[1] - P1[1]) * (P2[1] - P1[1]) +
                    (P2[2] - P1[2]) * (P2[2] - P1[2])
            );

            int steps_P1P2 = static_cast<int>(std::floor(distance_P1P2 / interval));
            float step_length_P1P2 = steps_P1P2 > 0 ? distance_P1P2 / steps_P1P2 : distance_P1P2;

            printf("\ndistance: %f steps: %i length: %f \n", distance_P1P2, steps_P1P2, step_length_P1P2);

            float diff_x = P2[0] - P1[0];
            float diff_y = P2[1] - P1[1];
            float diff_z = P2[2] - P1[2];

            float interval_x = steps_P1P2 > 0 ? diff_x / steps_P1P2 : 0;
            float interval_y = steps_P1P2 > 0 ? diff_y / steps_P1P2 : 0;
            float interval_z = steps_P1P2 > 0 ? diff_z / steps_P1P2 : 0;

            printf("\nSUBPATH\n");
            for (int i = 1; i < steps_P1P2; i++) {
                printf("x: %f y: %f z:%f\n", P1[0] + interval_x * i, P1[1] + interval_y * i, P1[2] + interval_z * i);

                printf("distance since last: %f\n", step_length_P1P2);



                int pointX = static_cast<int>(std::round(P1[0] + interval_x * i));
                int pointY = static_cast<int>(std::round(P1[1] + interval_y * i));


                printf("X: %i, Y: %i \n", pointX, pointY);

                underground = heightMap[pointY][pointX] >= P1[2] + interval_z * i;
                if(underground && undergroundLast) {
                    d_ug += step_length_P1P2;
                } else if (underground != undergroundLast) {
                    d_ug += step_length_P1P2 / 2;
                }
                undergroundLast = underground;



            }
            printf("x: %f y: %f z:%f\n", P2[0], P2[1], P2[2]);
            printf("distance since last: %f\n", step_length_P1P2);

            int p2X = static_cast<int>(std::round(P2[0]));
            int p2Y = static_cast<int>(std::round(P2[1]));

            underground = heightMap[p2Y][p2X] >= P2[2];
            if(underground && undergroundLast) {
                d_ug += step_length_P1P2;
            } else if (underground != undergroundLast) {
                d_ug += step_length_P1P2 / 2;
            }
            undergroundLast = underground;

        }
        printf("total underground  distance: %f\n", d_ug);

        //Penaly term P
        float P = d_ug + d_dz + d_ea + (N_wp_unsmoothed / N_wp * l_traj);

        //Fitness function F
        float F;
        if(P == 0) {
            F = 1 + 1 / (1+ 0/*C*/);
        } else {
            F = 0 + 1 / (1 + P);
        }



        fitnesses.push_back(F);

    }


    return fitnesses;
}

std::vector <std::vector<float>> computeFPA(
        const std::vector <std::vector<double>> &heightMap,
        int iter_max,
        int population,
        float p_switch,
        float epsilon_init,
        float epsilon_final,
        int two_opt_freq) {

    size_t x_mid = heightMap.size() / 2;
    size_t y_mid = (heightMap.empty()) ? 0 : heightMap[0].size() / 2;

    float x_min = 0.0f;
    float x_max = heightMap.size();
    float y_min = 0.0f;
    float y_max = heightMap[0].size();
    float z_min = -500.0f;
    float z_max = 500.0f;
    float x1 = 5.0f;
    float y1 = (float) y_mid;
    float z1 = heightMap[y1][x1] + 10;
    float xn = heightMap.size() - 5.0f;
    float yn = (float) y_mid;
    float zn = heightMap[yn][xn] + 10;



    //Objective min or max f(x), x = (x1, x2, ..., xd)
    //Initialize a population of n flowers/pollen gametes with random solutions
    std::vector <std::vector<float>> outputVector;
    for (int i = 0; i < population; i++) {
        outputVector.push_back(generateVector(x1, y1, z1, xn, yn, zn, x_min, x_max, y_min, y_max, z_min, z_max));
    }

    std::vector <std::vector<float>> smoothedPaths;
    smoothedPaths = smoothPaths(outputVector, 12.0f, 52);

    std::vector <std::vector<float>> testPaths;

    std::vector<float> test;

    test.push_back(10);
    test.push_back(10);
    test.push_back(1000);

    test.push_back(10);
    test.push_back(10);
    test.push_back(-1000);

    testPaths.push_back(test);

    std::vector<float> test3;

    test3.push_back(10);
    test3.push_back(140);
    test3.push_back(1000);

    test3.push_back(10);
    test3.push_back(140);
    test3.push_back(-1000);

    testPaths.push_back(test3);

    std::vector<float> test1;

    test1.push_back(140);
    test1.push_back(10);
    test1.push_back(1000);

    test1.push_back(140);
    test1.push_back(10);
    test1.push_back(-1000);

    testPaths.push_back(test1);

    std::vector<float> test2;

    test2.push_back(140);
    test2.push_back(140);
    test2.push_back(1000);

    test2.push_back(140);
    test2.push_back(140);
    test2.push_back(-1000);

    testPaths.push_back(test2);


    std::vector<float> fitnesses;
    fitnesses = computeFitnesses(smoothedPaths, heightMap);
    for (float fitness: fitnesses) {
        printf("\nfitness: %f", fitness);
    }
    printf("\n");

    save_to_csv(fitnesses, "../heightMapper/fitnesses.csv");


    //Find the best solution g∗ in the initial population

    //while (t <MaxGeneration)
    //   for i = 1 : n (all n flowers in the population)
    //if rand < p,
    //           Draw a (d-dimensional) step vector L which obeys a L´evy distribution
    //Global pollination via x

    //  else
    //  Draw ǫ from a uniform distribution in [0,1]
    //  Randomly choose j and k among all the solutions
    // Do local pollination via x

    //   end if
    //      Evaluate new solutions
    //  If new solutions are better, update them in the population
    //  end for
    //  Find the current best solution g∗
    //  end while

    return smoothedPaths;
}

