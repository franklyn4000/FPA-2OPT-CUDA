#include "hip/hip_runtime.h"
//
// Created by franklyn on 3/20/24.
//

#include "fitnessComputer.cuh"
#include <iostream>

std::vector<float> computeFitnesses(
        std::vector <std::vector<float>> paths,
        const std::vector <std::vector<double>> &heightMap,
        std::vector<float> N_wps, float max_asc_angle,
        float max_desc_angle, float a_utopia, float f_utopia) {

    std::vector<float> fitnesses;

    //penalty term

    //P = d_ug + d_dz + d_ea + (N_wp_unsmoothed / N_wp * l_traj)
    float w1 = 0.5;
    float w2 = 0.5;

    float f_cum = 0;


    float interval = 20.0f;

    for (const std::vector<float> &path: paths) {
        int n = path.size() / 3;

        for (int i = 0; i < n - 0; i++) {
            printf("\nP X: %f Y: %f Z: %f", path[3 * i], path[3 * i + 1], path[3 * i + 2]);
        }

    }

    int index = 0;
    for (const std::vector<float> &path: paths) {
        float d_ug = 0.0;
        float d_dz = 0.0;
        float d_ea = 0.0;
        float N_wp = 0.0;
        float l_traj = 0.0;


        float a_avg = 0;
        float f_avg = 0;

        int n = path.size() / 3;
        bool underground = false;
        bool undergroundLast = false;

        for (int i = 0; i < n - 1; i++) {
            std::vector<float> P1;
            std::vector<float> P2;
            P1.push_back(path[3 * i]);
            P1.push_back(path[3 * i + 1]);
            P1.push_back(path[3 * i + 2]);
            P2.push_back(path[3 * (i + 1)]);
            P2.push_back(path[3 * (i + 1) + 1]);
            P2.push_back(path[3 * (i + 1) + 2]);

            float distance_P1P2 = sqrt(
                    (P2[0] - P1[0]) * (P2[0] - P1[0]) +
                    (P2[1] - P1[1]) * (P2[1] - P1[1]) +
                    (P2[2] - P1[2]) * (P2[2] - P1[2])
            );

            int steps_P1P2 = static_cast<int>(std::floor(distance_P1P2 / interval));
            float step_length_P1P2 = steps_P1P2 > 0 ? distance_P1P2 / steps_P1P2 : distance_P1P2;


            printf("\ndistance: %f steps: %i length: %f \n", distance_P1P2, steps_P1P2, step_length_P1P2);

            float diff_x = P2[0] - P1[0];
            float diff_y = P2[1] - P1[1];
            float diff_z = P2[2] - P1[2];

            float interval_x = steps_P1P2 > 0 ? diff_x / steps_P1P2 : 0;
            float interval_y = steps_P1P2 > 0 ? diff_y / steps_P1P2 : 0;
            float interval_z = steps_P1P2 > 0 ? diff_z / steps_P1P2 : 0;


            double horizontal_length = std::sqrt(diff_x * diff_x + diff_y * diff_y);

            double angle_radians = std::atan2(diff_z, horizontal_length);


            printf("\nSUBPATH\n");
            for (int i = 1; i < steps_P1P2; i++) {
                printf("x: %f y: %f z:%f\n", P1[0] + interval_x * i, P1[1] + interval_y * i, P1[2] + interval_z * i);

                printf("distance since last: %f\n", step_length_P1P2);

                printf("DEBUG %f\n", P1[0] + interval_x * i);
                printf("DEBUG %f\n", P1[1] + interval_y * i);

                int pointX = static_cast<int>(std::round(P1[0] + interval_x * i));
                int pointY = static_cast<int>(std::round(P1[1] + interval_y * i));


                printf("X: %i, Y: %i \n", pointX, pointY);

                underground = heightMap[pointY][pointX] >= P1[2] + interval_z * i;
                if (underground && undergroundLast) {
                    d_ug += step_length_P1P2;
                } else if (underground != undergroundLast) {
                    d_ug += step_length_P1P2 / 2;
                }
                undergroundLast = underground;
                l_traj += step_length_P1P2;

            }
            printf("x: %f y: %f z:%f\n", P2[0], P2[1], P2[2]);
            printf("distance since last: %f\n", step_length_P1P2);

            int p1X = static_cast<int>(std::round(P1[0]));
            int p1Y = static_cast<int>(std::round(P1[1]));

            int p2X = static_cast<int>(std::round(P2[0]));
            int p2Y = static_cast<int>(std::round(P2[1]));

            int p1Z = static_cast<int>(std::round(P1[2]));
            int p2Z = static_cast<int>(std::round(P2[2]));

            underground = heightMap[p2Y][p2X] >= P2[2];
            if (underground && undergroundLast) {
                d_ug += step_length_P1P2;
            } else if (underground != undergroundLast) {
                d_ug += step_length_P1P2 / 2;
            }
            undergroundLast = underground;
            l_traj += step_length_P1P2;

            float height1 = p1Z - heightMap[p1Y][p1X];
            float height2 = p2Z - heightMap[p2Y][p2X];

            f_cum += (height1 + height2) / 2;

            printf("current angle %f°\n", angle_radians * (180.0 / M_PI));

            if (angle_radians > max_asc_angle || angle_radians < max_desc_angle) {
                d_ea += distance_P1P2;
            }


        }
        printf("total excessive flight path distance: %f\n", d_ea);
        printf("total underground  distance: %f\n", d_ug);

        N_wp = N_wps[index];
        //Penaly term P
        float P = d_ug + d_dz + d_ea + (N_wp * l_traj);

        f_avg = f_cum / n;

        //Cost term C
        float C = w1 * (l_traj / a_utopia) + w2 * (f_avg / f_utopia);

        //Fitness function F
        float F;
        if (P == 0) {
            F = 1 + 1 / (1 + C);
        } else {
            F = 0 + 1 / (1 + P);
        }


        fitnesses.push_back(F);
        index++;
    }


    return fitnesses;
}